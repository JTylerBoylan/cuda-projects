#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <chrono>
#include <assert.h>
#include <ginac/ginac.h>

#define TPB 256

using namespace GiNaC;

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

int main()
{

    symbol x("x");
    ex poly = pow(x,2) + 3*x + 2;

    std::cout << "Polynomial: " << poly << std::endl;

    return 0;
}