#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <chrono>
#include <assert.h>

#include "GENERATED_LOOKUP.cu"

#define NUM_ITERATIONS 50
#define TOLERANCE 1E-2

inline hipError_t checkCuda(hipError_t result);

__device__
float squared_sum(float * w)
{
  float sum = 0;
  for (int v = 0; v < NUM_VARIABLES; v++)
  {
    sum += w[v]*w[v];
  }
  return sum;
}

__global__
void solve(float * w, float * coeffs, float * cost, bool * sol)
{
  assert(NUM_VARIABLES == blockDim.x);

  // Variables local to the block
  __shared__ float wi[NUM_VARIABLES];
  __shared__ float diffi[NUM_VARIABLES];
  __shared__ bool solved;

  // Indices
  const int varIdx = threadIdx.x;
  const int objIdx = blockIdx.x;
  const int globalIdx = objIdx*NUM_VARIABLES + varIdx;

  // Initialize
  if (varIdx == 0)
  {
    solved = false;
  }
  __syncthreads();

  // Get from global lookup function
  wi[varIdx] = LOOKUP_INITIAL[globalIdx];

  // Run Newton-Raphson
  for (int iter = 0; iter < NUM_ITERATIONS; iter++)
  {

    // Break if solved
    if (solved) break;

    // Evaluate from global lookup function
    diffi[varIdx] = LOOKUP_INTERCEPT[globalIdx](wi, &coeffs[objIdx]);

    // Apply
    wi[varIdx] -= diffi[varIdx];

    // Check if solved
    if (varIdx == 0 && squared_sum(diffi) < TOLERANCE)
    {
      solved = true;
    }

    // Make sure the entire block is done before iterating
    __syncthreads();
  }

  // Save results
  w[globalIdx] = wi[varIdx];
  if (varIdx == 0)
  {
    cost[objIdx] = LOOKUP_OBJECTIVE[objIdx](wi, &coeffs[objIdx]);
    sol[objIdx] = solved;
  }

}

__global__
void generate_coefficients(const int size, float * coeffs)
{
  const int index = blockDim.x*blockIdx.x + threadIdx.x;

  const float maxVal = 0.5F;
  const float minVal = 0.25F;

  float coeffVal = minVal + (float(index)/float(size))*(maxVal - minVal);

  coeffs[index] = index % 2 == 0 ? coeffVal : 1 - coeffVal;
}

#define CYCLES 10000L

int main()
{

  // Allocate
  float * w;
  float * coeffs;
  float * cost;
  bool * sol;
  checkCuda( hipMallocManaged(&w, NUM_OBJECTIVES*NUM_VARIABLES*sizeof(float)) );
  checkCuda( hipMallocManaged(&coeffs, NUM_OBJECTIVES*NUM_COEFFICIENTS*sizeof(float)) );
  checkCuda( hipMallocManaged(&cost, NUM_OBJECTIVES*sizeof(float)) );
  checkCuda( hipMallocManaged(&sol, NUM_OBJECTIVES*sizeof(bool)) );

  // Solve
  auto cstart = std::chrono::high_resolution_clock::now();
  for (int cyc = 0; cyc < CYCLES; cyc++)
  {
    generate_coefficients<<<NUM_OBJECTIVES, NUM_COEFFICIENTS>>>(NUM_OBJECTIVES*NUM_COEFFICIENTS, coeffs);
    solve<<<NUM_OBJECTIVES, NUM_VARIABLES>>>(w, coeffs, cost, sol);
    checkCuda( hipDeviceSynchronize() );
  }
  auto cend = std::chrono::high_resolution_clock::now();

  time_t time_us = std::chrono::duration_cast<std::chrono::microseconds>(cend - cstart).count();
  printf("Cycles: %lu, Time: %lu us\n", CYCLES, time_us);
  printf("Performance: %lu cycles/s\n", CYCLES*(1000000L)/time_us);

  // Print Results
  for (int i = 0; i < NUM_OBJECTIVES*NUM_VARIABLES; i++)
  {
    printf("W(%d) = %f ", i, w[i]);
    printf("%s ", sol[i/NUM_VARIABLES] ? "(solved)" : "(unsolved)");
    printf("cost = %f\n", cost[i/NUM_VARIABLES]);
  }

  // Get Error
  const int NUM_STATES = 2;
  float solution[NUM_STATES] = {1.0, 0.0};
  float squared_sum_err = 0.0f;
  for (int p = 0; p < NUM_OBJECTIVES; p++)
  {
    int wIdx = p*NUM_VARIABLES;
    for (int s = 0; s < NUM_STATES; s++)
    {
      squared_sum_err += sqrt((w[wIdx+s] - solution[s])*(w[wIdx+s] - solution[s]));
    }
  }
  printf("Error: %f\n", squared_sum_err/NUM_OBJECTIVES);

  // Free
  checkCuda( hipFree(w) );
  checkCuda( hipFree(cost) );
  checkCuda( hipFree(sol) );

  return 0;
}

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}