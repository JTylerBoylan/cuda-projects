#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <chrono>
#include <assert.h>

#include "GENERATED_LOOKUP.cu"

#define NUM_ITERATIONS 50
#define TOLERANCE 1E-4

inline hipError_t checkCuda(hipError_t result);

__device__
float squared_sum(float * w)
{
  float sum = 0;
  for (int v = 0; v < NUM_VARIABLES; v++)
  {
    sum += w[v]*w[v];
  }
  return sum;
}

__global__
void solve(float * w, float * cost, bool * sol)
{
  assert(NUM_VARIABLES == blockDim.x);

  // Variables local to the block
  __shared__ float wi[NUM_VARIABLES];
  __shared__ float diffi[NUM_VARIABLES];
  __shared__ bool solved;

  // Indices
  const int varIdx = threadIdx.x;
  const int objIdx = blockIdx.x;
  const int globalIdx = objIdx*NUM_VARIABLES + varIdx;

  // Initialize
  if (varIdx == 0)
  {
    solved = false;
  }
  __syncthreads();

  // Get from global lookup function
  wi[varIdx] = LOOKUP_INITIAL[globalIdx];

  // Run Newton-Raphson
  for (int iter = 0; iter < NUM_ITERATIONS; iter++)
  {

    // Break if solved
    if (solved) break;

    // Evaluate from global lookup function
    diffi[varIdx] = LOOKUP_INTERCEPT[globalIdx](wi);

    // Apply
    wi[varIdx] -= diffi[varIdx];

    // Check if solved
    if (varIdx == 0 && squared_sum(diffi) < TOLERANCE)
    {
      solved = true;
    }

    // Make sure the entire block is done before iterating
    __syncthreads();
  }

  // Save results
  w[globalIdx] = wi[varIdx];
  if (varIdx == 0)
  {
    cost[objIdx] = COST(wi);
    sol[objIdx] = solved;
  }

}

#define CYCLES 10000L

int main()
{

  // Allocate
  float * w;
  float * cost;
  bool * sol;
  checkCuda( hipMallocManaged(&w, NUM_OBJECTIVES*NUM_VARIABLES*sizeof(float)) );
  checkCuda( hipMallocManaged(&cost, NUM_OBJECTIVES*sizeof(float)) );
  checkCuda( hipMallocManaged(&sol, NUM_OBJECTIVES*sizeof(bool)) );

  // Solve
  auto cstart = std::chrono::high_resolution_clock::now();
  for (int cyc = 0; cyc < CYCLES; cyc++)
  {
    solve<<<NUM_OBJECTIVES, NUM_VARIABLES>>>(w, cost, sol);
    checkCuda( hipDeviceSynchronize() );
  }
  auto cend = std::chrono::high_resolution_clock::now();

  time_t time_us = std::chrono::duration_cast<std::chrono::microseconds>(cend - cstart).count();
  printf("Cycles: %lu, Time: %lu us\n", CYCLES, time_us);
  printf("Performance: %lu cycles/s\n", CYCLES*(1000000L)/time_us);

  float solution[] = {1.0, 0.0, 0.0, 0.0, 2.0, 0.0, 0.0, 0.0};

  // Print Results
  float squared_sum_err = 0.0f;
  for (int i = 0; i < NUM_OBJECTIVES*NUM_VARIABLES; i++)
  {
    squared_sum_err += (w[i] - solution[i])*(w[i] - solution[i]);
    printf("W(%d) = %f ", i, w[i]);
    printf("%s ", sol[i/NUM_VARIABLES] ? "(solved)" : "(unsolved)");
    printf("cost = %f\n", cost[i/NUM_VARIABLES]);
  }
  printf("Error: %f\n", squared_sum_err);

  // Free
  checkCuda( hipFree(w) );
  checkCuda( hipFree(cost) );
  checkCuda( hipFree(sol) );

  return 0;
}

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}