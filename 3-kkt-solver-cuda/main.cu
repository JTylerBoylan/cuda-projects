
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <chrono>
#include <assert.h>

#define TPB 256

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

int main()
{

    return 0;
}