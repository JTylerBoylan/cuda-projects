#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <chrono>
#include <assert.h>
#include <hipsolver.h>
#include <hipblas.h>

#include "GENERATED_LOOKUP.cu"

#define NUM_ITERATIONS 50
#define TOLERANCE 1E-2

inline hipError_t checkCuda(hipError_t result);

#define CYCLES 1L

int main()
{

    float * w = 0;
    checkCuda( hipMalloc(&w, NUM_OBJECTIVES*NUM_VARIABLES*sizeof(float)) );

    float * coeffs = 0;
    checkCuda( hipMalloc(&coeffs, NUM_OBJECTIVES*NUM_COEFFICIENTS*sizeof(float)) );

    float ** KKT = 0;
    float * d_KKT = 0;
    checkCuda( hipMalloc(&KKT, NUM_OBJECTIVES*sizeof(float*)) );
    checkCuda( hipMalloc(&d_KKT, NUM_OBJECTIVES*NUM_VARIABLES*sizeof(float)) );

    float ** J = 0;
    float * d_J = 0;
    checkCuda( hipMalloc(&J, NUM_OBJECTIVES*sizeof(float*)) );
    checkCuda( hipMalloc(&d_J, NUM_OBJECTIVES*NUM_VARIABLES*NUM_VARIABLES*sizeof(float)) );

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int *d_infoArray;  // Info array
    int *d_PivotArray;  // Pivot array for LU factorization
    checkCuda( hipMalloc((void**)&d_infoArray, NUM_OBJECTIVES * sizeof(int)) );
    checkCuda( hipMalloc((void**)&d_PivotArray, NUM_VARIABLES * NUM_OBJECTIVES * sizeof(int)) );

    // Solve
    auto cstart = std::chrono::high_resolution_clock::now();
    for (int cyc = 0; cyc < CYCLES; cyc++)
    {

        GET_WI(w);

        // GET_COEFFS(coeffs);

        for (int iter = 0; iter < NUM_ITERATIONS; iter++)
        {
          GET_KKT(d_KKT, w, coeffs);
          FORMAT_KKT(KKT, d_KKT);

          GET_J(d_J, w, coeffs);
          FORMAT_J(J, d_J);

          checkCuda( hipDeviceSynchronize() );

          // LU Factorization
          hipblasSgetrfBatched(handle, NUM_VARIABLES, J, NUM_VARIABLES, d_PivotArray, d_infoArray, NUM_OBJECTIVES);

          // Solve
          hipblasSgetrsBatched(handle, HIPBLAS_OP_N, NUM_VARIABLES, NUM_OBJECTIVES, (const float**) J, NUM_VARIABLES,
            d_PivotArray, KKT, NUM_VARIABLES, d_infoArray, NUM_OBJECTIVES);


          
        }

    }
    auto cend = std::chrono::high_resolution_clock::now();

    time_t time_us = std::chrono::duration_cast<std::chrono::microseconds>(cend - cstart).count();
    printf("Cycles: %lu, Time: %lu us\n", CYCLES, time_us);
    printf("Performance: %lu cycles/s\n", CYCLES*(1000000L)/time_us);

    hipblasDestroy(handle);
    checkCuda( hipFree(w) );
    checkCuda( hipFree(coeffs) );
    checkCuda( hipFree(d_KKT) );
    checkCuda( hipFree(KKT) );
    checkCuda( hipFree(d_J) );
    checkCuda( hipFree(J) );

    return EXIT_SUCCESS;
}


inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}